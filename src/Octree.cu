#include "hip/hip_runtime.h"
#include "Octree.hpp"
#include "dequeue.hpp"
#include "MortonUtils.hpp"
#include "PointCloud.hpp"
#include "CudaCommon.hpp"
#include "cub/device/device_scan.cuh"

#include <vector>
#include <type_traits>

using namespace OT;
using hipcub::DeviceScan;

#define SEARCH_Q_SIZE (32)

__global__ void decodePoints(
    Point* points,
    const Code_t* codes,
    const float min_coord,
    const float range,
    const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        points[i] = codeToPoint(codes[i], min_coord, range);
    }
}

__device__ void OTNode::setChild(const int child, const int my_child_idx) {
    children[my_child_idx] = child;
    // atomic version of child_mask |= (1 << my_child_idx);
    atomicOr(&child_node_mask, 1 << my_child_idx);
}

__device__ void OTNode::setLeaf(const int leaf, const int my_child_idx) {
    children[my_child_idx] = leaf;
    // atomic version of child_mask &= ~(1 << my_child_idx);
    atomicOr(&child_leaf_mask, 1 << my_child_idx);
}

__global__ void calcEdgeCounts(
    const std::remove_pointer<decltype(RT::Nodes::prefixN)>::type *prefixN,
    const std::remove_pointer<decltype(RT::Nodes::parent)>::type *parents,
    int* rt_edge_counts,
    const size_t N) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    // root has no parent, so don't do for index 0
    if (i > 0 && i < N) {
        int my_depth = prefixN[i] / 3;
        int parent_depth = prefixN[parents[i]] / 3;
        rt_edge_counts[i] = my_depth - parent_depth;
    }
}

// __global__ void initializeOTNodes(
//     OTNode* nodes,
//     const int N) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if (i < N) {
//         nodes[i].child_mask = 0;
//         #pragma unroll
//         for (int child = 0; child < 8; ++child) {
//             nodes[i].children[child] = -1;
//         }
//     }
// }

__global__ void linkLeafNodes(
    OTNode* nodes,
    const int* node_offsets,
    const int* rt_node_counts,
    const Code_t* codes,
    const bool* rt_hasLeafLeft,
    const bool* rt_hasLeafRight,
    const uint8_t* rt_prefixN,
    const int* rt_leftChild,
    const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        // int n_new_nodes = rt_node_counts[i];
        // if (n_new_nodes > 0) {
            // link leaves if possible
            if (rt_hasLeafLeft[i]) {
                int leaf_idx = rt_leftChild[i];
                int leaf_level = rt_prefixN[i]/3 + 1;
                Code_t leaf_prefix = codes[leaf_idx] >> (CODE_LEN - (3 * leaf_level));
                int child_idx = leaf_prefix & 0b111;
                // link leaf to bottom octree node in string
                int bottom_oct = node_offsets[i];
                nodes[bottom_oct].setLeaf(leaf_idx, child_idx);
            }
            if (rt_hasLeafRight[i]) {
                int leaf_idx = rt_leftChild[i] + 1;
                int leaf_level = rt_prefixN[i]/3 + 1;
                Code_t leaf_prefix = codes[leaf_idx] >> (CODE_LEN - (3 * leaf_level));
                int child_idx = leaf_prefix & 0b111;
                int bottom_oct = node_offsets[i];
                nodes[bottom_oct].setLeaf(leaf_idx, child_idx);
            }
        // }
    }

}

__global__ void makeNodes(
    OTNode* nodes,
    const int* node_offsets,
    const int* rt_node_counts,
    const Code_t* codes,
    const uint8_t* rt_prefixN,
    const int* rt_parents,
    const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // TODO: What to do with node 0?
    if (i > 0 && i < N) {
        int oct_idx = node_offsets[i];
        // int n_new_nodes = node_offsets[i] - node_offsets[i - 1];
        int n_new_nodes = rt_node_counts[i];
        for (int j = 0; j < n_new_nodes - 1; ++j) {
            int level = rt_prefixN[i]/3 - j;
            Code_t node_prefix = codes[i] >> (CODE_LEN - (3 * level));
            int child_idx = node_prefix & 0b111;
            int parent = oct_idx + 1;
            nodes[parent].setChild(oct_idx, child_idx);
            oct_idx = parent;
        }
        if (n_new_nodes > 0) {
            int rt_parent = rt_parents[i];
            while (rt_node_counts[rt_parent] == 0) {
                rt_parent = rt_parents[rt_parent];
            }
            int oct_parent = node_offsets[rt_parent];
            int top_level = rt_prefixN[i]/3 - n_new_nodes + 1;
            Code_t top_node_prefix = codes[i] >> (CODE_LEN - (3 * top_level));
            int child_idx = top_node_prefix & 0b111;
            nodes[oct_parent].setChild(oct_idx, child_idx);
        }
    }
}

Octree::Octree(const RT::RadixTree& radix_tree) {
    // Number of octree nodes between a node and its parent
    int* rt_edge_counts;
    CudaCheckCall(hipMallocManaged(&rt_edge_counts, sizeof(*rt_edge_counts) * radix_tree.n_nodes));
    // Copy a "1" to the first element to account for the root
    rt_edge_counts[0] = 1;
    int blocks, tpb;
    std::tie(blocks, tpb) = makeLaunchParams(radix_tree.n_nodes);
    calcEdgeCounts<<<blocks, tpb>>>(radix_tree.d_tree.prefixN, radix_tree.d_tree.parent, rt_edge_counts, radix_tree.n_nodes);
	hipDeviceSynchronize();
    CudaCheckError();

    // Inclusive prefix sum to find location of each octree node
    int* oc_node_offsets;
    CudaCheckCall(hipMallocManaged(&oc_node_offsets, (1 + radix_tree.n_nodes) * sizeof(*oc_node_offsets)));
    oc_node_offsets[0] = 0;

    void* d_temp_storage = nullptr;
    size_t temp_storage_reqd = 0;
    CudaCheckCall(
        DeviceScan::InclusiveSum(d_temp_storage, temp_storage_reqd,
                rt_edge_counts, oc_node_offsets + 1,
                radix_tree.n_nodes)
    );
    CudaCheckCall(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_reqd));
    CudaCheckCall(
        DeviceScan::InclusiveSum(d_temp_storage, temp_storage_reqd,
                rt_edge_counts, oc_node_offsets + 1,
                radix_tree.n_nodes)
    );
	hipDeviceSynchronize();
    CudaCheckError();
    g_allocator.DeviceFree(d_temp_storage);

    auto n_oct_nodes = oc_node_offsets[radix_tree.n_nodes];
    printf("total nodes: %d\n", n_oct_nodes);

    CudaCheckCall(hipMallocManaged(&nodes, n_oct_nodes * sizeof(*nodes)));

    // setup initial values of octree node objects
    CudaCheckCall(hipMemset(nodes, 0, n_oct_nodes * sizeof(*nodes)));
    // std::tie(blocks, tpb) = makeLaunchParams(n_oct_nodes);
    // initializeOTNodes<<<blocks, tpb>>>(nodes, n_oct_nodes);

    std::tie(blocks, tpb) = makeLaunchParams(radix_tree.n_nodes);
    makeNodes<<<blocks, tpb>>>(nodes,
                               oc_node_offsets,
                               rt_edge_counts,
                               radix_tree.d_tree.mortonCode,
                               radix_tree.d_tree.prefixN,
                               radix_tree.d_tree.parent,
                               radix_tree.n_nodes);

    linkLeafNodes<<<blocks, tpb>>>(nodes,
                                   oc_node_offsets,
                                   rt_edge_counts,
                                   radix_tree.d_tree.mortonCode,
                                   radix_tree.d_tree.hasLeafLeft,
                                   radix_tree.d_tree.hasLeafRight,
                                   radix_tree.d_tree.prefixN,
                                   radix_tree.d_tree.leftChild,
                                   radix_tree.n_nodes);

    // hipDeviceSynchronize();
    // for (int i = 0; i < n_oct_nodes; ++i) {
    //     printf("Node %d:\n\tparent: %d\n\tchildren:\n", i, nodes[i].parent);
    //     for (int j = 0; j < 8; ++j) {
    //         if (nodes[i].child_node_mask & (1 << j)) {
    //             printf("\t\tNode %d: %d\n", j, nodes[i].children[j]);
    //         }
    //         if (nodes[i].child_leaf_mask & (1 << j)) {
    //             printf("\t\tLeaf %d: %d\n", j, nodes[i].children[j]);
    //         }
    //     }
    // }

    // free temporary memory from construction
    CudaCheckCall(hipFree(rt_edge_counts));
    CudaCheckCall(hipFree(oc_node_offsets));

    // decode points for use later
    CudaCheckCall(hipMallocManaged(&u_points, radix_tree.n_pts * sizeof(Point)));
    std::tie(blocks, tpb) = makeLaunchParams(radix_tree.n_pts);
    decodePoints<<<blocks, tpb>>>(u_points,
                                   radix_tree.d_tree.mortonCode,
                                   radix_tree.min_coord,
                                   radix_tree.max_coord - radix_tree.min_coord,
                                   radix_tree.n_pts);

    hipDeviceSynchronize();
    CudaCheckError();
}

Octree::~Octree() {
    CudaCheckCall(hipFree(nodes));
}

template <int k>
__global__ void knnSearch(
    const OTNode* octree,
    const Point* query_pts,
    Point* result_pts,
    const float eps,
    const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        const Point query = query_pts[i];
        PriorityQueue<OTNode, SEARCH_Q_SIZE> queue;
        // distance explored so far
        float r = 0;
        // best distance so far
        float d = INFINITY;
        while (d >= (1 + eps) * r) {
            // if 
        }

    }
}

template <int k>
std::vector<std::array<Point, k>> knnSearch(const std::vector<Point>& points) {
    const int n = points.size();
    std::vector<std::array<Point, k>> results;
    // results in unified memory. each array of k elements is stored back-to-back
    Point* d_results;
    // query points;
    Point* d_query;
    // allocate device storage 
    CudaCheckCall(hipMallocManaged(&d_results, n * k * sizeof(Point)));
    CudaCheckCall(hipMalloc(&d_query, n * sizeof(Point)));
    // transfer query points to device memory
    CudaCheckCall(hipMemcpy(d_query, &points[0], n * sizeof(Point), hipMemcpyHostToDevice));

    int blocks, tpb;
    std::tie(blocks, tpb) = makeLaunchParams(n);
    knnSearch<k><<<blocks, tpb>>>(d_query, d_results, 0.1, n);
    hipDeviceSynchronize();
    CudaCheckError();

    results.resize(points.size());
    CudaCheckCall(hipMemcpy(&results[0], d_results, n * k * sizeof(Point)));


    CudaCheckCall(hipFree(d_query));
    CudaCheckCall(hipFree(d_results));

    return results;
}