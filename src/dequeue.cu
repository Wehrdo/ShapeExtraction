#include "hip/hip_runtime.h"
#include "dequeue.hpp"

/*
////////////////////////////////////////////////////////////////////////
--------------------Priority Queue Implementation-----------------------
////////////////////////////////////////////////////////////////////////
*/

__host__ __device__ int int_log2(int x) {
    int ret = 0;
    while (x >>= 1) ++ret;
    return ret;
}

template <typename T, int MAX_Q>
__host__ __device__ void PriorityQueue<T, MAX_Q>::insert(const T* data, float weight) {
    //assert(size < MAX_Q); // Must not overfill array
    //Q_Node newN = { node, weight };

    // If full, replace the current highest with this, only if it is lower
    if (size == MAX_Q) {
        if (peekMax().priority >= weight) {
            removeMax();
        }
        else {
            return;
        }
    }
    size += 1;
    int idx = size - 1;
    data[idx] = { data, weight };
    bubbleUp(idx);
    //while (idx > 0) {
    //	int parent_idx = parent(idx);
    //	if (data[parent_idx].priority > weight) {
    //		data[idx] = data[parent_idx];
    //		idx = parent_idx;
    //	}
    //	else {
    //		break;
    //	}
    //}
    //data[idx] = { node, weight };
}

template <typename T, int MAX_Q>
__host__ __device__ void PriorityQueue<T, MAX_Q>::swap(int idx1, int idx2) {
    Q_Node<T> original_idx1_val = data[idx1];
    data[idx1] = data[idx2];
    data[idx2] = original_idx1_val;
}

template <typename T, int MAX_Q>
__host__ __device__ void PriorityQueue<T, MAX_Q>::bubbleUp(int idx) {
    if (int_log2(idx + 1) % 2 == 0) {
        if (parent(idx) >= 0 && data[idx].priority > data[parent(idx)].priority) {
            swap(idx, parent(idx));
            bubbleUpMax(parent(idx));
        }
        else {
            bubbleUpMin(idx);
        }
    }
    else {
        if (parent(idx) >= 0 && data[idx].priority < data[parent(idx)].priority) {
            swap(idx, parent(idx));
            bubbleUpMin(parent(idx));
        }
        else {
            bubbleUpMax(idx);
        }
    }
}

template <typename T, int MAX_Q>
__host__ __device__ void PriorityQueue<T, MAX_Q>::bubbleUpMin(int idx) {
    // While idx has a grandparent
    while (idx >= 3) {
        int grandparent_idx = parent(parent(idx));
        if (data[idx].priority < data[grandparent_idx].priority) {
            swap(idx, grandparent_idx);
            idx = grandparent_idx;
        }
        else {
            break;
        }
    }
}

template <typename T, int MAX_Q>
__host__ __device__ void PriorityQueue<T, MAX_Q>::bubbleUpMax(int idx) {
    // While idx has a grandparent
    while (idx >= 3) {
        int grandparent_idx = parent(parent(idx));
        if (data[idx].priority > data[grandparent_idx].priority) {
            swap(idx, grandparent_idx);
            idx = grandparent_idx;
        }
        else {
            break;
        }
    }
}

template <typename T, int MAX_Q>
__host__ __device__ Q_Node<T> PriorityQueue<T, MAX_Q>::removeMin() {
    //assert(size > 0); // Canot remove from empty queue
    Q_Node<T> to_return = data[0];
    size--;
    data[0] = data[size];
    percolateDown(data[size], 0);
    return to_return;
}

template <typename T, int MAX_Q>
__host__ __device__ int PriorityQueue<T, MAX_Q>::largest_idx() {
    int largest;
    if (size >= 2) {
        if (size == 2) {
            // Only 2 nodes, so maximum is the only node on level 1 (the first max level)
            largest = 1;
        }
        else {
            // Largest of left or right on level 1
            if (data[1].priority > data[2].priority) {
                largest = 1;
            }
            else { largest = 2; }
        }
    }
    else { largest = 0; } // Only one node
    return largest;
}

template <typename T, int MAX_Q>
__host__ __device__ Q_Node<T> PriorityQueue<T, MAX_Q>::removeMax() {
    int largest = largest_idx();
    Q_Node<T> to_return = data[largest];
    size--;
    data[largest] = data[size];
    percolateDown(data[size], largest);
    return to_return;
}

template <typename T, int MAX_Q>
__host__ __device__ Q_Node<T> PriorityQueue<T, MAX_Q>::peekMax() {
    int largest = largest_idx();
    return data[largest];
}

template <typename T, int MAX_Q>
__host__ __device__ void PriorityQueue<T, MAX_Q>::percolateDown(Q_Node<T> item, int idx) {
    if (int_log2(idx + 1) % 2 == 0) {
        // Even, min-level
        percolateDownMin(item, idx);
    }
    else {
        // Odd, max-level
        percolateDownMax(item, idx);
    }
    //// If smallest child is less than item
    //if (data[smallestChild].priority < item.priority) {
    //	// Move smallest child up
    //	data[idx] = data[smallestChild];
    //	idx = smallestChild;
    //}
    //// Otherwise done
    //else {
    //	break;
    //}
}

template <typename T, int MAX_Q>
__host__ __device__ void PriorityQueue<T, MAX_Q>::percolateDownMin(Q_Node<T> item, int idx) {
    // While node at idx has children
    while (childLeft(idx) < size) {
        // Find the smallest among children and grandchildren
        int descendents[] = {/*childLeft(idx), */childRight(idx),
            childLeft(childLeft(idx)), childRight(childLeft(idx)),
            childLeft(childRight(idx)), childRight(childRight(idx)) };
        int smallestChild = childLeft(idx);
        for (int desc_idx : descendents) {
            if (desc_idx < size && data[desc_idx].priority < data[smallestChild].priority) {
                smallestChild = desc_idx;
            }
        }

        if (smallestChild > childRight(idx)) {
            // smallest is a grandchild
            if (data[smallestChild].priority < data[idx].priority) {
                swap(idx, smallestChild);
                if (data[smallestChild].priority > data[parent(smallestChild)].priority) {
                    swap(parent(smallestChild), smallestChild);
                }
                idx = smallestChild;
            }
            else {
                break;
            }
        }
        else {
            // smallest is a child
            if (data[smallestChild].priority < data[idx].priority) {
                swap(idx, smallestChild);
                idx = smallestChild;
            }
            break;
        }
    }
}

template <typename T, int MAX_Q>
__host__ __device__ void PriorityQueue<T, MAX_Q>::percolateDownMax(Q_Node<T> item, int idx) {
    // While node at idx has children
    while (childLeft(idx) < size) {
        // Find the smallest among children and grandchildren
        int descendents[] = {/*childLeft(idx), */childRight(idx),
            childLeft(childLeft(idx)), childRight(childLeft(idx)),
            childLeft(childRight(idx)), childRight(childRight(idx)) };
        int smallestChild = childLeft(idx);
        for (int desc_idx : descendents) {
            if (desc_idx < size && data[desc_idx].priority > data[smallestChild].priority) {
                smallestChild = desc_idx;
            }
        }

        if (smallestChild > childRight(idx)) {
            // smallest is a grandchild
            if (data[smallestChild].priority > data[idx].priority) {
                swap(idx, smallestChild);
                if (data[smallestChild].priority < data[parent(smallestChild)].priority) {
                    swap(parent(smallestChild), smallestChild);
                }
                idx = smallestChild;
            }
            else {
                break;
            }
        }
        else {
            // smallest is a child
            if (data[smallestChild].priority > data[idx].priority) {
                swap(idx, smallestChild);
                idx = smallestChild;
            }
            break;
        }
    }
}