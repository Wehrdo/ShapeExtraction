#include "hip/hip_runtime.h"
#include "RadixTree.hpp"
#include "CudaCommon.cuh"
#include "libmorton/include/morton.h"
#include "cub/device/device_reduce.cuh"
#include "cub/device/device_radix_sort.cuh"

#include <array>
#include <algorithm>
#include <limits>
#include <cstdint>
#include <type_traits>

#include <math.h>

using namespace RT;
using hipcub::DeviceReduce;
using hipcub::DeviceRadixSort;

template <typename T>
__global__ void makeCodes(const T minCoord,
                          const T maxCoord,
                          const T* __restrict__ x_vals,
                          const T* __restrict__ y_vals,
                          const T* __restrict__ z_vals,
                          Node* nodes,
                          const size_t N) {
    // only supports 1-dimension blocks and grids
    assert(threadIdx.y == threadIdx.z == 1);
    assert(blockIdx.y == blockIdx.z == 1);

    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        T range = (maxCoord - minCoord);
        // We can only encode 21 bits (21 <bits> * 3 <dimensions> = 63 <bits>)
        const uint32_t bitscale = 0xFFFFFFFFu >> (32 - 21);
        uint32_t x_coord = bitscale * ((x_vals[idx] - minCoord) / range);
        uint32_t y_coord = bitscale * ((y_vals[idx] - minCoord) / range);
        uint32_t z_coord = bitscale * ((z_vals[idx] - minCoord) / range);
        nodes[idx].mortonCode = morton3D_64_encode(x_coord, y_coord, z_coord);
        // if (idx == 0) {
        //     // printf("min = %f, max = %f\n", minCoord, maxCoord);
        //     printf("%u, %u, %u\n", x_coord, y_coord, z_coord);
        //     printf("%f, %f, %f = %x\n", x_vals[idx], y_vals[idx], z_vals[idx], nodes[idx].mortonCode);

        //     uint_fast32_t dec_raw_x, dec_raw_y, dec_raw_z;
        //     morton3D_64_decode(nodes[idx].mortonCode, dec_raw_x, dec_raw_y, dec_raw_z);
        //     float dec_x = ((float)dec_raw_x / bitscale) * range + minCoord;
        //     float dec_y = ((float)dec_raw_y / bitscale) * range + minCoord;
        //     float dec_z = ((float)dec_raw_z / bitscale) * range + minCoord;
        //     printf("decoded = %f, %f, %f\n", dec_x, dec_y, dec_z);
        // }
    }
}

__global__ void fillCodes(const Node* nodes, Code_t* codes, const size_t N) {
    assert(threadIdx.y == threadIdx.z == 1);
    assert(blockIdx.y == blockIdx.z == 1);

    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        codes[idx] = nodes[idx].mortonCode;
    }
}

// computes ceil(a / b)
template<typename T>
__device__ inline T ceil_div(T a, T b) {
    // If a + b might overflow, do the following instead? (untested):
    //     1 + ((x - 1) / y); // if x != 0
    assert(!std::is_signed<decltype(a)>() || a >= 0);
    assert(!std::is_signed<decltype(b)>() || b >= 0);
    return (a + b - 1) / b;
}

// delta(a, b) is the length of the longest prefix between codes a and b
__device__ inline uint_fast8_t delta(const Code_t a, const Code_t b) {
    // Assuming first bit is 0. Asserts check that.
    // Not necessary, so if want to store info in that bit in the future, requires a change
    Code_t bit1_mask = (Code_t)1 << (sizeof(a) * 8 - 1);
    assert(a & bit1_mask == 0);
    assert(b & bit1_mask == 0);
    return __clzll(a ^ b) - 1;
}

__global__ void constructTree(Node* nodes, const size_t N) {
    assert(threadIdx.y == threadIdx.z == 1);
    assert(blockIdx.y == blockIdx.z == 1);

    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        auto code_i = nodes[i].mortonCode;
        // Determine direction of the range (+1 or -1)
        // TODO: This will break when i = 0 or i = n-1
        auto delta_diff_right = delta(code_i, nodes[i+1].mortonCode);
        auto delta_diff_left = delta(code_i, nodes[i-1].mortonCode);
        int_fast8_t direction_difference = delta_diff_right - delta_diff_left;
        int_fast8_t d = (direction_difference > 0) - (direction_difference < 0);
        assert(d == -1 || d == 1);

        // Compute upper bound for the length of the range
        auto delta_min = delta(code_i, nodes[i - d].mortonCode);
        Code_t l_max = 2;
        // Cast to ptrdiff_t so in case the result is negative (since d is +/- 1), we can catch it and not index out of bounds
        while (static_cast<ptrdiff_t>(i) + static_cast<ptrdiff_t>(l_max)*d >= 0 &&
               i + l_max*d < N &&
               delta(code_i, nodes[i + l_max * d].mortonCode) > delta_min) {
            l_max *= 2;
        }
        // Find the other end using binary search
        Code_t l = 0;
        uint_fast8_t divisor;
        size_t t;
        for (t = l_max / 2, divisor = 2; t >= 1; divisor *= 2, t = l_max / divisor) {
            if (delta(code_i, nodes[i + (l + t)*d].mortonCode) > delta_min) {
                l += t;
            }
        }
        size_t j = i + l*d;
        // Find the split position using binary search
        auto delta_node = delta(nodes[i].mortonCode, nodes[j].mortonCode);
        size_t s = 0;
        for (t = ceil_div<Code_t>(l, 2), divisor = 2; t >= 1; divisor *= 2, t = ceil_div<Code_t>(l, divisor)) {
            if (delta(code_i, nodes[i + (s + t)*d].mortonCode) > delta_node) {
                s += t;
            }
        }

        // Split position
        size_t gamma = i + s*d + min(d, 0);
        nodes[i].leftChild = gamma;
        nodes[i].hasLeafLeft = (min(i, j) == gamma);
        nodes[i].hasLeafRight = (max(i, j) == gamma+1);
        // Set parents of left and right children, if they aren't leaves
        // can't set this node as parent of its leaves, because the
        // leaf also represents an internal node with a differnent parent
        if (!nodes[i].hasLeafLeft) {
            nodes[gamma].parent = i;
        }
        if (!nodes[i].hasLeafRight) {
            nodes[gamma + 1].parent = i;
        }
    }
}

 std::tuple<int, int> makeLaunchParams(size_t n, int tpb = 512) {
    // int tpb = 256;
    int blocks = (n + tpb - 1) / tpb;
    return std::make_tuple(blocks, tpb);
 }

void RadixTree::encodePoints(const PointCloud<float>& cloud) {
    // Check that the cast is okay
    assert(cloud.x_vals.size() <= std::numeric_limits<decltype(n_pts)>::max());
    n_pts = static_cast<decltype(n_pts)>(cloud.x_vals.size());

    // Allocate for tree
    size_t tree_size = n_pts * sizeof(Node);
    CudaCheckCall(hipMalloc(&d_tree, tree_size));
    // Allocate for raw data points
    size_t data_size = n_pts * sizeof(cloud.x_vals[0]);
    float *d_data_x, *d_data_y, *d_data_z;
    CudaCheckCall(hipMalloc(&d_data_x, data_size));
    CudaCheckCall(hipMalloc(&d_data_y, data_size));
    CudaCheckCall(hipMalloc(&d_data_z, data_size));
    // Copy points to GPU
    CudaCheckCall(hipMemcpyAsync(d_data_x, &cloud.x_vals[0], data_size, hipMemcpyHostToDevice));
    CudaCheckCall(hipMemcpyAsync(d_data_y, &cloud.y_vals[0], data_size, hipMemcpyHostToDevice));
    CudaCheckCall(hipMemcpyAsync(d_data_z, &cloud.z_vals[0], data_size, hipMemcpyHostToDevice));
    hipDeviceSynchronize();

    // Find maximum and minumum values in data
    std::array<float, 3> mins, maxes;
    float *d_mins, *d_maxes;
    CudaCheckCall(g_allocator.DeviceAllocate((void**)&d_mins, sizeof(float) * 3));
    CudaCheckCall(g_allocator.DeviceAllocate((void**)&d_maxes, sizeof(float) * 3));

    size_t temp_storage_reqd = 0;
    void* d_temp_storage = nullptr;
    // get amount of required memory
    DeviceReduce::Max(d_temp_storage, temp_storage_reqd, d_data_x, &maxes[0], n_pts);
    // allocate temporary storage
    CudaCheckCall(g_allocator.DeviceAllocate((void**)&d_temp_storage,  temp_storage_reqd));
    // Find maximum
    DeviceReduce::Max(d_temp_storage, temp_storage_reqd, d_data_x, &d_maxes[0], n_pts);
    DeviceReduce::Max(d_temp_storage, temp_storage_reqd, d_data_y, &d_maxes[1], n_pts);
    DeviceReduce::Max(d_temp_storage, temp_storage_reqd, d_data_z, &d_maxes[2], n_pts);
    DeviceReduce::Min(d_temp_storage, temp_storage_reqd, d_data_x, &d_mins[0], n_pts);
    DeviceReduce::Min(d_temp_storage, temp_storage_reqd, d_data_y, &d_mins[1], n_pts);
    DeviceReduce::Min(d_temp_storage, temp_storage_reqd, d_data_z, &d_mins[2], n_pts);
    hipDeviceSynchronize();
    CudaCheckError();

    hipMemcpy(&mins[0], d_mins, sizeof(float) * mins.size(), hipMemcpyDeviceToHost);
    hipMemcpy(&maxes[0], d_maxes, sizeof(float) * maxes.size(), hipMemcpyDeviceToHost);
    g_allocator.DeviceFree(d_mins);
    g_allocator.DeviceFree(d_maxes);
    g_allocator.DeviceFree(d_temp_storage);
    hipDeviceSynchronize();
    float max_val = *std::max_element(maxes.begin(), maxes.end());
    float min_val = *std::min_element(mins.begin(), mins.end());
    // std::cout << "range = [" << min_val << ", " << max_val << "]" << std::endl;

    int blocks, tpb;
    std::tie(blocks, tpb) = makeLaunchParams(n_pts);
    makeCodes<<<blocks, tpb>>>(min_val, max_val, d_data_x, d_data_y, d_data_z, d_tree, n_pts);
    hipDeviceSynchronize();
    CudaCheckError();

    // Now that codes created, raw values not needed
    CudaCheckCall(hipFree(d_data_x));
    CudaCheckCall(hipFree(d_data_y));
    CudaCheckCall(hipFree(d_data_z));
}

RadixTree::RadixTree(const PointCloud<float>& cloud) {
    // fill up mortonCode in d_tree
    encodePoints(cloud);

    // Sort in ascending order
    // Just the Morton codes from the nodes
    Code_t *d_keys;
    CudaCheckCall(hipMalloc(&d_keys, sizeof(*d_keys) * n_pts));
    int blocks, tpb;
    std::tie(blocks, tpb) = makeLaunchParams(n_pts);
    fillCodes<<<blocks, tpb>>>(d_tree, d_keys, n_pts);
    hipDeviceSynchronize();
    CudaCheckError();
    void* d_temp_storage = nullptr;
    size_t temp_storage_reqd = 0;
    CudaCheckCall(
        // get storage requirements
        DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_reqd,
                                   d_keys, d_keys,
                                   d_tree, d_tree,
                                   n_pts)
    );
    CudaCheckCall(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_reqd));
    // sort key-value pairs, where key is morton code (d_keys), and values are tree nodes
    CudaCheckCall(
        DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_reqd,
                                   d_keys, d_keys,
                                   d_tree, d_tree,
                                   n_pts)
    );
    hipDeviceSynchronize();
    CudaCheckError();
    g_allocator.DeviceFree(d_temp_storage);

    // Make tree

}

RadixTree::~RadixTree() {
    CudaCheckCall(hipFree(d_tree));
}
