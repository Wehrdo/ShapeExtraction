#include "hip/hip_runtime.h"
#include "RadixTree.hpp"
#include "CudaCommon.hpp"
#include "libmorton/include/morton.h"
#include "cub/device/device_reduce.cuh"
#include "cub/device/device_radix_sort.cuh"

#include <memory>
#include <array>
#include <algorithm>
#include <limits>
#include <cstdint>
#include <type_traits>

#include <math.h>

using namespace RT;
using hipcub::DeviceReduce;
using hipcub::DeviceRadixSort;

template <typename T>
__global__ void makeCodes(const T minCoord,
                          const T maxCoord,
                          const T* __restrict__ x_vals,
                          const T* __restrict__ y_vals,
                          const T* __restrict__ z_vals,
                          Code_t* codes,
                          const size_t N) {
    // only supports 1-dimension blocks and grids
    assert(threadIdx.y == threadIdx.z == 1);
    assert(blockIdx.y == blockIdx.z == 1);

    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        T range = (maxCoord - minCoord);
        // We can only encode 21 bits (21 <bits> * 3 <dimensions> = 63 <bits>)
        const uint32_t bitscale = 0xFFFFFFFFu >> (32 - (codeLen / 3));
        uint32_t x_coord = bitscale * ((x_vals[idx] - minCoord) / range);
        uint32_t y_coord = bitscale * ((y_vals[idx] - minCoord) / range);
        uint32_t z_coord = bitscale * ((z_vals[idx] - minCoord) / range);
        codes[idx] = morton3D_64_encode(x_coord, y_coord, z_coord);
        // if (idx == 0) {
        //     // printf("min = %f, max = %f\n", minCoord, maxCoord);
        //     printf("%u, %u, %u\n", x_coord, y_coord, z_coord);
        //     printf("%f, %f, %f = %x\n", x_vals[idx], y_vals[idx], z_vals[idx], nodes[idx].mortonCode);

        //     uint_fast32_t dec_raw_x, dec_raw_y, dec_raw_z;
        //     morton3D_64_decode(nodes[idx].mortonCode, dec_raw_x, dec_raw_y, dec_raw_z);
        //     float dec_x = ((float)dec_raw_x / bitscale) * range + minCoord;
        //     float dec_y = ((float)dec_raw_y / bitscale) * range + minCoord;
        //     float dec_z = ((float)dec_raw_z / bitscale) * range + minCoord;
        //     printf("decoded = %f, %f, %f\n", dec_x, dec_y, dec_z);
        // }
    }
}

// __global__ void fillCodes(const Node* nodes, Code_t* codes, const size_t N) {
//     assert(threadIdx.y == threadIdx.z == 1);
//     assert(blockIdx.y == blockIdx.z == 1);

//     size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx < N) {
//         codes[idx] = nodes[idx].mortonCode;
//     }
// }

// computes ceil(a / b)
template<typename T>
__device__ inline T ceil_div(T a, T b) {
    // If a + b might overflow, do the following instead? (untested):
    //     1 + ((x - 1) / y); // if x != 0
    assert(!std::is_signed<decltype(a)>() || a >= 0);
    assert(!std::is_signed<decltype(b)>() || b >= 0);
    return (a + b - 1) / b;
}

__device__ inline int log2_ceil(Code_t x) {
    static_assert(sizeof(x) == sizeof(long long int), "__clzll(x) is for long long int");
    // Counting from LSB to MSB, number of bits before last '1'
    // This is floor(log(x))
    int n_lower_bits = (8 * sizeof(x)) - __clzll(x) - 1;
    // Add 1 if 2^n_lower_bits is less than x
    //     (i.e. we rounded down because x was not a power of 2)
    return n_lower_bits + (x > (1 << n_lower_bits));
}

// delta(a, b) is the length of the longest prefix between codes a and b
__device__ inline int_fast8_t delta(const Code_t a, const Code_t b) {
    // Assuming first bit is 0. Asserts check that.
    // Not necessary, so if want to store info in that bit in the future, requires a change
    Code_t bit1_mask = (Code_t)1 << (sizeof(a) * 8 - 1);
	//if (a & bit1_mask) {
	//	printf("omg");
	//}
    assert((a & bit1_mask) == 0);
    assert((b & bit1_mask) == 0);
    return __clzll(a ^ b) - 1;
}

__global__ void constructTree(const Code_t* codes,
                              bool* hasLeafLeft,
                              bool* hasLeafRight,
                              int* leftChild,
                              int* parent,
                              uint8_t* prefixN,
                              const size_t N) {
    assert(threadIdx.y == threadIdx.z == 1);
    assert(blockIdx.y == blockIdx.z == 1);

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        auto code_i = codes[i];
        // Determine direction of the range (+1 or -1)
        int d;
        if (i == 0) {
            d = 1;
        }
        else {
            auto delta_diff_right = delta(code_i, codes[i+1]);
            auto delta_diff_left = delta(code_i, codes[i-1]);
            int direction_difference = delta_diff_right - delta_diff_left;
            d = (direction_difference > 0) - (direction_difference < 0);
        }

        // Compute upper bound for the length of the range
        
        Code_t l = 0;
        if (i == 0) {
            // First node is root, covering whole tree
            l = N - 1;
        }
        else {
            auto delta_min = delta(code_i, codes[i - d]);
            Code_t l_max = 2;
            // Cast to ptrdiff_t so in case the result is negative (since d is +/- 1), we can catch it and not index out of bounds
            while (i + static_cast<ptrdiff_t>(l_max)*d >= 0 &&
                i + l_max*d < N &&
                delta(code_i, codes[i + l_max * d]) > delta_min) {
                l_max *= 2;
            }
            int t;
            int divisor;
            // Find the other end using binary search
            for (t = l_max / 2, divisor = 2; t >= 1; divisor *= 2, t = l_max / divisor) {
                if (delta(code_i, codes[i + (l + t)*d]) > delta_min) {
                    l += t;
                }
            }
        }
        int j = i + l*d;
        // Find the split position using binary search
        auto delta_node = delta(codes[i], codes[j]);
        prefixN[i] = delta_node;
        int s = 0;
        int t;
        int max_divisor = 1 << log2_ceil(l);
        int divisor = 2;
        for (t = ceil_div<Code_t>(l, 2); divisor <= max_divisor; divisor <<= 1, t = ceil_div<Code_t>(l, divisor)) {
        // for (t = ceil_div<Code_t>(l, 2), divisor = 2; t >= 1; divisor *= 2, t = ceil_div<Code_t>(l, divisor)) {
            if (delta(code_i, codes[i + (s + t)*d]) > delta_node) {
                s += t;
            }
        }

        // Split position
        int gamma = i + s*d + min(d, 0);
        leftChild[i] = gamma;
        hasLeafLeft[i] = (min(i, j) == gamma);
        hasLeafRight[i] = (max(i, j) == gamma+1);
        // Set parents of left and right children, if they aren't leaves
        // can't set this node as parent of its leaves, because the
        // leaf also represents an internal node with a differnent parent
        if (!hasLeafLeft[i]) {
            parent[gamma] = i;
        }
        if (!hasLeafRight[i]) {
            parent[gamma + 1] = i;
        }
    }
}

void RadixTree::encodePoints(const PointCloud<float>& cloud) {
    // Allocate for raw data points
    size_t data_size = n_pts * sizeof(cloud.x_vals[0]);
    float *d_data_x, *d_data_y, *d_data_z;
    CudaCheckCall(hipMalloc(&d_data_x, data_size));
    CudaCheckCall(hipMalloc(&d_data_y, data_size));
    CudaCheckCall(hipMalloc(&d_data_z, data_size));
    // Copy points to GPU
    CudaCheckCall(hipMemcpyAsync(d_data_x, &cloud.x_vals[0], data_size, hipMemcpyHostToDevice));
    CudaCheckCall(hipMemcpyAsync(d_data_y, &cloud.y_vals[0], data_size, hipMemcpyHostToDevice));
    CudaCheckCall(hipMemcpyAsync(d_data_z, &cloud.z_vals[0], data_size, hipMemcpyHostToDevice));
    hipDeviceSynchronize();

    // Find maximum and minumum values in data
    std::array<float, 3> mins, maxes;
    float *d_mins, *d_maxes;
    CudaCheckCall(g_allocator.DeviceAllocate((void**)&d_mins, sizeof(float) * 3));
    CudaCheckCall(g_allocator.DeviceAllocate((void**)&d_maxes, sizeof(float) * 3));

    size_t temp_storage_reqd = 0;
    void* d_temp_storage = nullptr;
    // get amount of required memory
    DeviceReduce::Max(d_temp_storage, temp_storage_reqd, d_data_x, &maxes[0], n_pts);
    // allocate temporary storage
    CudaCheckCall(g_allocator.DeviceAllocate((void**)&d_temp_storage,  temp_storage_reqd));
    // Find maximum
    DeviceReduce::Max(d_temp_storage, temp_storage_reqd, d_data_x, &d_maxes[0], n_pts);
    DeviceReduce::Max(d_temp_storage, temp_storage_reqd, d_data_y, &d_maxes[1], n_pts);
    DeviceReduce::Max(d_temp_storage, temp_storage_reqd, d_data_z, &d_maxes[2], n_pts);
    DeviceReduce::Min(d_temp_storage, temp_storage_reqd, d_data_x, &d_mins[0], n_pts);
    DeviceReduce::Min(d_temp_storage, temp_storage_reqd, d_data_y, &d_mins[1], n_pts);
    DeviceReduce::Min(d_temp_storage, temp_storage_reqd, d_data_z, &d_mins[2], n_pts);
    hipDeviceSynchronize();
    CudaCheckError();

    hipMemcpy(&mins[0], d_mins, sizeof(float) * mins.size(), hipMemcpyDeviceToHost);
    hipMemcpy(&maxes[0], d_maxes, sizeof(float) * maxes.size(), hipMemcpyDeviceToHost);
    g_allocator.DeviceFree(d_mins);
    g_allocator.DeviceFree(d_maxes);
    g_allocator.DeviceFree(d_temp_storage);
    hipDeviceSynchronize();
    float max_val = *std::max_element(maxes.begin(), maxes.end());
    float min_val = *std::min_element(mins.begin(), mins.end());
    // std::cout << "range = [" << min_val << ", " << max_val << "]" << std::endl;

    int blocks, tpb;
    std::tie(blocks, tpb) = makeLaunchParams(n_pts);
    makeCodes<<<blocks, tpb>>>(min_val, max_val, d_data_x, d_data_y, d_data_z, d_tree.mortonCode, n_pts);
    hipDeviceSynchronize();
    CudaCheckError();

    // Now that codes created, raw values not needed
    CudaCheckCall(hipFree(d_data_x));
    CudaCheckCall(hipFree(d_data_y));
    CudaCheckCall(hipFree(d_data_z));
}

RadixTree::RadixTree(const PointCloud<float>& cloud) {
    // Check that the cast is okay
    assert(cloud.x_vals.size() <= std::numeric_limits<decltype(n_pts)>::max());
    n_pts = static_cast<decltype(n_pts)>(cloud.x_vals.size());
    // allocate memory for tree
    CudaCheckCall(hipMallocManaged(&d_tree.mortonCode, sizeof(*d_tree.mortonCode) * n_pts));
    CudaCheckCall(hipMallocManaged(&d_tree.hasLeafLeft, sizeof(*d_tree.hasLeafRight) * n_pts));
    CudaCheckCall(hipMallocManaged(&d_tree.hasLeafRight, sizeof(*d_tree.hasLeafRight) * n_pts));
    CudaCheckCall(hipMallocManaged(&d_tree.prefixN, sizeof(*d_tree.prefixN) * n_pts));
    CudaCheckCall(hipMallocManaged(&d_tree.leftChild, sizeof(*d_tree.leftChild) * n_pts));
    CudaCheckCall(hipMallocManaged(&d_tree.parent, sizeof(*d_tree.parent) * n_pts));

    // fill up mortonCode in d_tree
    encodePoints(cloud);

    // Sort in ascending order
    // Just the Morton codes from the nodes
    Code_t* d_codes_sorted;
    CudaCheckCall(hipMalloc(&d_codes_sorted, sizeof(*d_codes_sorted) * n_pts));
    void* d_temp_storage = nullptr;
    size_t temp_storage_reqd = 0;
    CudaCheckCall(
        // get storage requirements
        DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_reqd,
                                  d_tree.mortonCode, d_codes_sorted,
                                  n_pts)
    );
    CudaCheckCall(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_reqd));
    // sort key-value pairs, where key is morton code (d_keys), and values are tree nodes
    CudaCheckCall(
        DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_reqd,
                                   d_tree.mortonCode, d_codes_sorted,
                                   n_pts)
    );
    hipDeviceSynchronize();
    CudaCheckError();
    g_allocator.DeviceFree(d_temp_storage);
    // TODO: Remove duplicates
    n_nodes = n_pts - 1;

    // Swap out keys for sorted keys
    // Okay to do at this point, because nothing else in d_tree has been filled
    CudaCheckCall(hipFree(d_tree.mortonCode));
    d_tree.mortonCode = d_codes_sorted;

    // Make tree
    int blocks, tpb;
    std::tie(blocks, tpb) = makeLaunchParams(n_nodes);
    constructTree<<<blocks, tpb>>>(d_tree.mortonCode,
                                   d_tree.hasLeafLeft,
                                   d_tree.hasLeafRight,
                                   d_tree.leftChild,
                                   d_tree.parent,
                                   d_tree.prefixN,
                                   n_nodes);
	hipDeviceSynchronize();
    CudaCheckError();

    // Code_t* h_codes = new Code_t[n_pts]();
    // CudaCheckCall(hipMemcpy(h_codes, d_tree.mortonCode, sizeof(Code_t) * n_pts, hipMemcpyDeviceToHost));
    // //auto h_leftChild = d_tree.mortonCode;
    // //auto h_parent = d_tree.parent;
    // auto h_leftChild = new std::remove_pointer<decltype(d_tree.leftChild)>::type[n_pts];
    // auto h_parent = new std::remove_pointer<decltype(d_tree.parent)>::type[n_pts];
    // //Code_t* h_leftChild = new std::remove_pointer<decltype(d_tree.mortonCode)>::type[n_pts]();
    // CudaCheckCall(hipMemcpy(h_codes, d_tree.mortonCode, sizeof(*h_codes) * n_pts, hipMemcpyDeviceToHost));
    // CudaCheckCall(hipMemcpy(h_leftChild, d_tree.leftChild, sizeof(*h_leftChild) * n_pts, hipMemcpyDeviceToHost));
    // CudaCheckCall(hipMemcpy(h_parent, d_tree.parent, sizeof(*h_parent) * n_pts, hipMemcpyDeviceToHost));
    // for (int i = 0; i < n_pts; ++i) {
    //     // std::cout << std::hex << h_tree[i].mortonCode << ", left = " << h_tree[i].leftChild << ",parent = " << h_tree[i].parent << std::endl;
    //     printf("idx = %d, code = %llx, left = %d, parent = %d\n",
    //             i, h_codes[i], h_leftChild[i], h_parent[i]);
    // }
}

RadixTree::~RadixTree() {
    CudaCheckCall(hipFree(d_tree.mortonCode));
    CudaCheckCall(hipFree(d_tree.hasLeafLeft));
    CudaCheckCall(hipFree(d_tree.hasLeafRight));
    CudaCheckCall(hipFree(d_tree.prefixN));
    CudaCheckCall(hipFree(d_tree.leftChild));
    CudaCheckCall(hipFree(d_tree.parent));
}
